#include "gpu_utils.cuh"


hipblasStatus_t templateGemm(hipblasHandle_t handle,
   hipblasOperation_t transa, hipblasOperation_t transb,
   int m, int n, int k,
   const float           *alpha,
   const float           *A, int lda,
   const float           *B, int ldb,
   const float           *beta,
   float           *C, int ldc) {
   return hipblasSgemm(handle,
   transa, transb,
   m, n, k,
   alpha,
   A, lda,
   B, ldb,
   beta,
   C, ldc);
}

hipblasStatus_t templateGemm(hipblasHandle_t handle,
   hipblasOperation_t transa, hipblasOperation_t transb,
   int m, int n, int k,
   const double           *alpha,
   const double           *A, int lda,
   const double           *B, int ldb,
   const double           *beta,
   double           *C, int ldc) {
   return hipblasDgemm(handle,
   transa, transb,
   m, n, k,
   alpha,
   A, lda,
   B, ldb,
   beta,
   C, ldc);
}

hiprandStatus_t templateCurandNormal(
    hiprandGenerator_t generator, 
    float *outputPtr, size_t n, 
    float mean, float stddev) {
    return hiprandGenerateNormal(generator, outputPtr, n, mean, stddev);
}

hiprandStatus_t templateCurandNormal(
    hiprandGenerator_t generator, 
    double *outputPtr, size_t n, 
    double mean, double stddev) {
    return hiprandGenerateNormalDouble(generator, outputPtr, n, mean, stddev);
}
