#include "hip/hip_runtime.h"
#include "integrator.cuh"
#include <ctime>
#include <iostream>
#include <vector>
#include <stdexcept>

void test_hvp() {

    timemachine::Integrator<float> intg;
    
    // const int N3 = 2500*3;
    // const int P = 500;

    const int N3 = 5;
    const int P = 3;

    float *A, *B, *C;
    A = (float*)malloc(N3*N3*sizeof(float));
    B = (float*)malloc(P*N3*sizeof(float));
    C = (float*)malloc(P*N3*sizeof(float));
 
    // Allocate device storage for A,B,C
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, N3*N3*sizeof(float));
    hipMalloc((void**)&d_B, P*N3*sizeof(float));
    hipMalloc((void**)&d_C, P*N3*sizeof(float));
 


    // Fill A,B diagonals with sin(i) data, C diagonal with cos(i)^2
    // Matrices are arranged column major
    for(size_t i=0; i<N3; i++) {
        for(size_t j=i; j<N3; j++) {

            if(i==j) {
                A[i*N3 + j] = sin(i*N3+j);
            } else {
                A[i*N3 + j] = cos(i*N3+j);
                A[j*N3 + i] = cos(i*N3+j);
            }
        }   
    }

    for(size_t i=0; i < N3; i++) {
        for(size_t j=0; j < N3; j++) {
            std::cout << A[i*N3+j] << " " ;
        }
        std::cout << std::endl;
    }
 
    for(size_t p=0; p<P; p++) {
        for(size_t j=0; j<N3; j++) {
            size_t index = p*N3 + j;
            if(p==j) {
                B[index] = sin(index);
                C[index] = cos(index)*cos(index);
            }
        else {
                B[index] = sin(index)*sin(index);
                C[index] = sin(index)*cos(index);;
            }
        }   
    }

    cublasErrchk(hipblasSetMatrix(N3, N3, sizeof(float), A, N3, d_A, N3));
    cublasErrchk(hipblasSetMatrix(N3, P, sizeof(float), B, N3, d_B, N3));
    cublasErrchk(hipblasSetMatrix(N3, P, sizeof(float), C, N3, d_C, N3));
 
    // reference implementation
    std::vector<float> expected(P*N3, 0);
    for(int p=0; p < P; p++) {
        for(int row=0; row < N3; row++) {
            float sum = 0;
            for(int col=0; col < N3; col++) {
                sum += A[row*N3 + col] * B[p*N3 + col];
            }
            expected[p*N3 + row] += sum + C[p*N3 + row];
        }
    }

    std::clock_t start;
    float duration;
    start = std::clock();
    intg.hessian_vector_product(d_A, d_B, d_C, N3, P);

    hipDeviceSynchronize();
    duration = ( std::clock() - start ) / (float) CLOCKS_PER_SEC;
    std::cout <<"speed: "<< duration/10 <<'\n';

    cublasErrchk(hipblasGetMatrix(N3, P, sizeof(float), d_C, N3, C, N3));

    for(int p=0; p < P; p++) {
        for(int row=0; row < N3; row++) {
            float ref = expected[p*N3 + row];
            float test = C[p*N3+row];
            if(fabs(ref - test) > 1e-3) {
                std::cout << ref << " " << test << std::endl;
                throw(std::runtime_error("FATAL"));
            }
        }
    }
}

void test_reduce() {

    const int W = 100;
    const int P = 500;
    const int N3 = 2500*3;

    // const int W = 10;
    // const int P = 5;
    // const int N3 = 12;



    float *WPN3 = (float*)malloc(W*P*N3*sizeof(float));
 
    for(size_t w=0; w < W; w++) {
        for(size_t p=0; p < P; p++) {
            for(size_t i=0; i<N3; i++) {
                WPN3[w*P*N3 + p*N3 + i] = sin(w+p+i);
            }
        }   
    }



    float *coeff_bs = (float*)malloc(P*N3*sizeof(float));
    for(size_t p=0; p < P; p++) {
        for(size_t i=0; i<N3; i++) {
            coeff_bs[p*N3 + i] = cos(p+i);
        }
    }

    // Allocate device storage for A,B,C
    float *d_WPN3;
    float *d_coeff_bs;

    gpuErrchk(hipMalloc((void**)&d_WPN3, W*P*N3*sizeof(float)));
    gpuErrchk(hipMalloc((void**)&d_coeff_bs, P*N3*sizeof(float)));

// __host__ ​hipError_t hipMemcpy ( void* dst, const void* src, size_t count, hipMemcpyKind kind ) 

    gpuErrchk(hipMemcpy(d_WPN3, WPN3, W*P*N3*sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_coeff_bs, coeff_bs, P*N3*sizeof(float), hipMemcpyHostToDevice));

    size_t tpb = 32;
    const size_t tot = P*N3;
    size_t n_blocks = (tot + tpb - 1) / tpb;

    float coeff_a = 0.9;

    int starting_k = 0;


    std::clock_t start;
    float duration;
    start = std::clock();

    // for(size_t i=0; i < 10; i++ ) {
        // std::cout << n_blocks << " " << tpb << std::endl;
        reduce_total<float><<<n_blocks, tpb>>>(
            coeff_a,
            d_coeff_bs,
            d_WPN3,
            starting_k,
            W,
            P*N3
        );
    // }

    gpuErrchk(hipDeviceSynchronize());



    duration = ( std::clock() - start ) / (float) CLOCKS_PER_SEC;
    std::cout <<"speed: "<< duration <<'\n';

    std::vector<float> test(P*N3);

    gpuErrchk(hipMemcpy(&test[0], d_WPN3, P*N3*sizeof(float), hipMemcpyDeviceToHost));

    std::vector<float> expected(P*N3);
    for(size_t p=0; p < P; p++) {
        for(size_t i=0; i<N3; i++) {
            float accum = 0;
            float prefactor = 0.0;
            float a_n = 1.0;
            for(size_t w=0; w < W; w++) {
                prefactor += a_n;
                a_n *= coeff_a;
                accum += prefactor*WPN3[w*P*N3 + p*N3 + i];
            }
            expected[p*N3 + i] = coeff_bs[p*N3 + i]*accum;
        }
    }

    for(size_t p=0; p < P; p++) {
        for(size_t i=0; i < N3; i++) {
            float a = test[p*N3+i];
            float b = expected[p*N3+i];
            if(fabs(a-b) > 1e-4) {
                throw(std::runtime_error("lol"));
            }

        }
    }

}

int main() {


    test_reduce();

}