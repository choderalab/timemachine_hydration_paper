#include "hip/hip_runtime.h"

#include "hipblas.h"

// #include <ctime>
#include <vector>
#include <stdexcept>
#include <cstdio>

#include "integrator.hpp"

#include "gpu_utils.cuh"


/*

Buffer operations:

Let W be the number of windows

1. Compute the total derivative Dx_t, using dxdp_t, hessians, and mixed partials
2. Add window_t into the converged window sums
3. Replace window_t with Dx_t
4. Reduce over all the windows and the converged buffer.
5. Update dx/dp_t to t+1

Each thread processes 1 out of [P,N,3] elements.

*/

template<typename NumericType>
__global__ void reduce_total(
    NumericType coeff_a,
    const NumericType *coeff_bs,
    const NumericType *Dx_t,
    NumericType *total_buffer,
    NumericType *converged_buffer,        
    NumericType *dxdp_t,
    int t, // starting window slot
    int W, // number of windows
    int PN3 // PN3
) {

    // 1. Done by SGEMM call
    int local_idx = blockIdx.x*blockDim.x + threadIdx.x;
    if(local_idx >= PN3) {
        return;
    }

    // 2. Add total_buffer[t] into converged buffer
    int window_idx = t * PN3 + blockIdx.x * blockDim.x + threadIdx.x;
    converged_buffer[local_idx] += total_buffer[window_idx];

    // 3. Replace window_t with Dx_t
    total_buffer[window_idx] = Dx_t[local_idx];

    // 4. Reduce over all the windows.
    NumericType prefactor = 0.0;
    NumericType a_n = 1.0;
    NumericType accum = 0.0;

    //      iter i
    // k=0  0 3 2 1
    // k=1  1 0 3 2
    // k=2  1 0 3 2
    // k=3  2 1 0 3
    for(int i=0; i < W; i++) {
        int slot = t - i < 0 ? t - i + W : t - i;
        int slot_idx = slot*PN3 + blockIdx.x*blockDim.x + threadIdx.x;
        prefactor += a_n;
        a_n *= coeff_a;
        // printf("%d %d %f\n", i, blockIdx.x*blockDim.x + threadIdx.x, total_buffer[slot_idx]);
        // printf("%d %d %f\n", i, blockIdx.x*blockDim.x + threadIdx.x, total_buffer[slot_idx]);

        if(local_idx == 0) {
            printf("w: %d, pref: %f, tot_buf: %f\n", i, prefactor, total_buffer[slot_idx]);
        }

        accum += prefactor*total_buffer[slot_idx];
    }

    // 5. Compute new dxdp_t
    // (ytz). coeff_b's can be optimized into smaller chunks.
    dxdp_t[local_idx] = -coeff_bs[local_idx] * (accum + prefactor * converged_buffer[local_idx]);

}

#include <iostream>
namespace timemachine {


template<typename NumericType> 
std::vector<NumericType> Integrator<NumericType>::get_dxdp() const {
    std::vector<NumericType> cpu_dxdp(P_*N_*3);
    gpuErrchk(hipMemcpy(&cpu_dxdp[0], d_dxdp_t_, P_*N_*3*sizeof(NumericType), hipMemcpyDeviceToHost));
    return cpu_dxdp;
}

template<typename NumericType> 
Integrator<NumericType>::Integrator(
    NumericType coeff_a,
    int W,
    int N,
    int P,
    const std::vector<NumericType> &coeff_bs) :
    W_(W),
    N_(N),
    P_(P),
    step_(0),
    coeff_a_(coeff_a) {

    if(coeff_bs.size() != N) {
        throw(std::runtime_error("Expected coeffbs to be PxNx3 shape"));
    }

    std::vector<NumericType> expanded_coeff_bs(P*N*3);
    for(size_t p=0; p < P; p++) {
        for(size_t n=0; n < N; n++) {
            for(size_t d=0; d < 3; d++) {
                expanded_coeff_bs[p*N*3+n*3+d] = coeff_bs[n];
            }
        }
    }

    // 1. Allocate memory on the GPU
    gpuErrchk(hipMalloc((void**)&d_dxdp_t_, P_*N_*3*sizeof(NumericType)));
    gpuErrchk(hipMalloc((void**)&d_total_buffer_, W_*P_*N_*3*sizeof(NumericType)));
    gpuErrchk(hipMalloc((void**)&d_converged_buffer_, P_*N_*3*sizeof(NumericType)));
    gpuErrchk(hipMalloc((void**)&d_coeff_bs_, P_*N_*3*sizeof(NumericType)));

    gpuErrchk(hipMalloc((void**)&d_hessians_, N_*3*N_*3*sizeof(NumericType)));
    gpuErrchk(hipMalloc((void**)&d_mixed_partials_, P_*N_*3*sizeof(NumericType)));
    // 2. Memset
    gpuErrchk(hipMemset(d_dxdp_t_, 0.0, P_*N_*3*sizeof(NumericType)));
    gpuErrchk(hipMemset(d_total_buffer_, 0.0, W_*P_*N_*3*sizeof(NumericType)));
    gpuErrchk(hipMemset(d_converged_buffer_, 0.0, P_*N_*3*sizeof(NumericType)));
    gpuErrchk(hipMemcpy(d_coeff_bs_, &expanded_coeff_bs[0], P_*N_*3*sizeof(NumericType), hipMemcpyHostToDevice));

    cublasErrchk(hipblasCreate(&cb_handle_));

}


template<typename NumericType> 
Integrator<NumericType>::~Integrator() {
    gpuErrchk(hipFree(d_dxdp_t_));
    gpuErrchk(hipFree(d_total_buffer_));
    gpuErrchk(hipFree(d_converged_buffer_));
    gpuErrchk(hipFree(d_coeff_bs_));

    gpuErrchk(hipFree(d_hessians_));
    gpuErrchk(hipFree(d_mixed_partials_));

    cublasErrchk(hipblasDestroy(cb_handle_));
}


// dangerous! not exception safe.
template<typename NumericType> 
void Integrator<NumericType>::step_cpu(
    const NumericType *h_hessians,
    const NumericType *h_mixed_partials) {

    gpuErrchk(hipMemcpy(d_hessians_, h_hessians, N_*3*N_*3*sizeof(NumericType), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_mixed_partials_, h_mixed_partials, P_*N_*3*sizeof(NumericType), hipMemcpyHostToDevice));

    step_gpu(d_hessians_, d_mixed_partials_);
}

template<typename NumericType> 
void Integrator<NumericType>::step_gpu(
    const NumericType *d_hessians,
    NumericType *d_mixed_partials) {

    hessian_vector_product(d_hessians_, d_dxdp_t_, d_mixed_partials);

    // std::vector<NumericType> debug(P_*N_*3);
    // gpuErrchk(hipMemcpy(&debug[0], d_mixed_partials, sizeof(float)*P_*N_*3, hipMemcpyDeviceToHost));
    // for(size_t i=0; i < P_*N_*3; i++) {
    //     std::cout << "hvp:" << debug[i] << " " << std::endl;
    // }

    std::cout << "STEPPING INTO: " << step_ % W_ << std::endl;

    reduce_buffers(d_mixed_partials_, step_ % W_);

    step_ += 1;

}

template<typename NumericType> 
void Integrator<NumericType>::reduce_buffers(const NumericType *d_Dx_t, int window_k) {

    size_t tpb = 32;
    const size_t tot = P_*N_*3;
    size_t n_blocks = (tot + tpb - 1) / tpb;

    reduce_total<NumericType><<<n_blocks, tpb>>>(
        coeff_a_,
        d_coeff_bs_,
        d_Dx_t,
        d_total_buffer_,
        d_converged_buffer_,
        d_dxdp_t_,
        window_k,
        W_,
        P_*N_*3
    );

    gpuErrchk(hipPeekAtLastError());

};

template<typename NumericType> 
void Integrator<NumericType>::hessian_vector_product(
    const NumericType *d_A,
    NumericType *d_B,
    NumericType *d_C) {

    NumericType alpha = 1.0;
    NumericType beta  = 1.0;
 
    const size_t N3 = N_*3;

    // replace with SGEMM later
    cublasErrchk(hipblasDgemm(cb_handle_,
        HIPBLAS_OP_N, HIPBLAS_OP_N, // whether or not we transpose A
        N3, P_, N3,
        &alpha,
        d_A, N3,
        d_B, N3,
        &beta,
        d_C, N3));
}

}

// template class timemachine::Integrator<float>;
template class timemachine::Integrator<double>;