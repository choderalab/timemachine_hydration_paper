#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "hiprand.h"

#include <iostream>
#include <vector>
#include <stdexcept>
#include <cstdio>

#include "integrator.hpp"
#include "gpu_utils.cuh"

template <typename NumericType>
__global__ void reduce_velocities(
    const NumericType *noise,
    const NumericType coeff_a,
    const NumericType *coeff_bs, // N x 3, not P x N x 3, but we could just pass in the first index
    const NumericType *coeff_cs,
    const NumericType *grads,
    const NumericType d_t,
    NumericType *x_t,
    NumericType *v_t,
    int N3) {

    int local_idx = blockIdx.x*blockDim.x + threadIdx.x;
    if(local_idx >= N3) {
        return;
    }

    v_t[local_idx] = coeff_a*v_t[local_idx] - coeff_bs[local_idx]*grads[local_idx] + coeff_cs[local_idx]*noise[local_idx];
    x_t[local_idx] += v_t[local_idx]*d_t;
}


template<typename NumericType>
__global__ void update_derivatives(
    NumericType coeff_a,
    const NumericType *coeff_bs,
    const NumericType *hmp,
    NumericType *dxdp_t,
    NumericType *dvdp_t,
    NumericType dt,
    int PN3 // num_params * num_atoms * 3
) {

    int local_idx = blockIdx.x*blockDim.x + threadIdx.x;
    if(local_idx >= PN3) {
        return;
    }

    NumericType tmp = coeff_a*dvdp_t[local_idx] - coeff_bs[local_idx]*hmp[local_idx];
    dvdp_t[local_idx] = tmp;
    dxdp_t[local_idx] += dt*tmp;

}


namespace timemachine {


template<typename NumericType> 
Integrator<NumericType>::Integrator(
    NumericType dt,
    int W,
    int N,
    int P,
    const NumericType coeff_a,
    const std::vector<NumericType> &coeff_bs,
    const std::vector<NumericType> &coeff_cs) :
    dt_(dt),
    W_(W),
    N_(N),
    P_(P),
    step_(0),
    coeff_a_(coeff_a) {

    if(coeff_bs.size() != N) {
        throw(std::runtime_error("Expected coeffbs to be PxNx3 shape"));
    }
    if(coeff_cs.size() != N) {
        throw(std::runtime_error("Expected coeffbs to be PxNx3 shape"));
    }

    std::vector<NumericType> expanded_coeff_bs(P*N*3);
    for(size_t p=0; p < P; p++) {
        for(size_t n=0; n < N; n++) {
            for(size_t d=0; d < 3; d++) {
                expanded_coeff_bs[p*N*3+n*3+d] = coeff_bs[n];
            }
        }
    }

    std::vector<NumericType> expanded_coeff_cs(N*3);
    for(size_t n=0; n < N; n++) {
        for(size_t d=0; d < 3; d++) {
            expanded_coeff_cs[n*3+d] = coeff_cs[n];
        }
    }

    // 1. Allocate memory on the GPU
    gpuErrchk(hipMalloc((void**)&d_x_t_, N_*3*sizeof(NumericType)));
    gpuErrchk(hipMalloc((void**)&d_v_t_, N_*3*sizeof(NumericType)));
    gpuErrchk(hipMalloc((void**)&d_dxdp_t_, P_*N_*3*sizeof(NumericType)));
    gpuErrchk(hipMalloc((void**)&d_dvdp_t_, P_*N_*3*sizeof(NumericType)));
    gpuErrchk(hipMalloc((void**)&d_coeff_bs_, P_*N_*3*sizeof(NumericType)));
    gpuErrchk(hipMalloc((void**)&d_coeff_cs_, N_*3*sizeof(NumericType)));

    // 2. Per-step buffers
    gpuErrchk(hipMalloc((void**)&d_energy_, sizeof(NumericType)));
    gpuErrchk(hipMalloc((void**)&d_grads_, N_*3*N_*3*sizeof(NumericType)));
    gpuErrchk(hipMalloc((void**)&d_hessians_, N_*3*N_*3*sizeof(NumericType)));
    gpuErrchk(hipMalloc((void**)&d_mixed_partials_, P_*N_*3*sizeof(NumericType)));
    gpuErrchk(hipMalloc((void**)&d_rng_buffer_, N_*3*sizeof(NumericType)));

    // 3. Memset
    gpuErrchk(hipMemset(d_x_t_, 0.0, N_*3*sizeof(NumericType)));
    gpuErrchk(hipMemset(d_v_t_, 0.0, N_*3*sizeof(NumericType)));
    gpuErrchk(hipMemset(d_dxdp_t_, 0.0, P_*N_*3*sizeof(NumericType)));
    gpuErrchk(hipMemset(d_dvdp_t_, 0.0, P_*N_*3*sizeof(NumericType)));

    gpuErrchk(hipMemcpy(d_coeff_bs_, &expanded_coeff_bs[0], P_*N_*3*sizeof(NumericType), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_coeff_cs_, &expanded_coeff_cs[0], N_*3*sizeof(NumericType), hipMemcpyHostToDevice));

    cublasErrchk(hipblasCreate(&cb_handle_));
    curandErrchk(hiprandCreateGenerator(&cr_rng_, HIPRAND_RNG_PSEUDO_PHILOX4_32_10));

    // (ytz): looks like by default cuRand always sets the default seed to 0.
    hiprandSetPseudoRandomGeneratorSeed(cr_rng_, time(NULL));

}


template <typename NumericType>
void Integrator<NumericType>::reset() {
    step_ = 0;
    gpuErrchk(hipMemset(d_x_t_, 0.0, N_*3*sizeof(NumericType)));
    gpuErrchk(hipMemset(d_v_t_, 0.0, N_*3*sizeof(NumericType)));
    gpuErrchk(hipMemset(d_dxdp_t_, 0.0, P_*N_*3*sizeof(NumericType)));
    gpuErrchk(hipMemset(d_dvdp_t_, 0.0, P_*N_*3*sizeof(NumericType)));


    gpuErrchk(hipMemset(d_energy_, 0, sizeof(NumericType)));
    gpuErrchk(hipMemset(d_grads_, 0, N_*3*sizeof(NumericType)));
    gpuErrchk(hipMemset(d_hessians_, 0, N_*3*N_*3*sizeof(NumericType)));
    gpuErrchk(hipMemset(d_mixed_partials_, 0, P_*N_*3*sizeof(NumericType)));


    hipDeviceSynchronize();
}


template<typename NumericType> 
Integrator<NumericType>::~Integrator() {
    gpuErrchk(hipFree(d_x_t_));
    gpuErrchk(hipFree(d_v_t_));
    gpuErrchk(hipFree(d_dxdp_t_));
    gpuErrchk(hipFree(d_dvdp_t_));
    gpuErrchk(hipFree(d_coeff_bs_));
    gpuErrchk(hipFree(d_coeff_cs_));

    gpuErrchk(hipFree(d_grads_));
    gpuErrchk(hipFree(d_hessians_));
    gpuErrchk(hipFree(d_mixed_partials_));
    gpuErrchk(hipFree(d_rng_buffer_));

    cublasErrchk(hipblasDestroy(cb_handle_));
    curandErrchk(hiprandDestroyGenerator(cr_rng_));
}

template<typename NumericType> 
std::vector<NumericType> Integrator<NumericType>::get_dxdp() const {
    std::vector<NumericType> buf(P_*N_*3);
    gpuErrchk(hipMemcpy(&buf[0], d_dxdp_t_, P_*N_*3*sizeof(NumericType), hipMemcpyDeviceToHost));
    return buf;
}

template<typename NumericType> 
std::vector<NumericType> Integrator<NumericType>::get_noise() const {
    std::vector<NumericType> buf(N_*3);
    gpuErrchk(hipMemcpy(&buf[0], d_rng_buffer_, N_*3*sizeof(NumericType), hipMemcpyDeviceToHost));
    return buf;
};

template<typename NumericType> 
std::vector<NumericType> Integrator<NumericType>::get_coordinates() const {
    std::vector<NumericType> buf(N_*3);
    gpuErrchk(hipMemcpy(&buf[0], d_x_t_, N_*3*sizeof(NumericType), hipMemcpyDeviceToHost));
    return buf;
};

template<typename NumericType> 
std::vector<NumericType> Integrator<NumericType>::get_velocities() const {
    std::vector<NumericType> buf(N_*3);
    gpuErrchk(hipMemcpy(&buf[0], d_v_t_, N_*3*sizeof(NumericType), hipMemcpyDeviceToHost));
    return buf;
};

template<typename NumericType> 
void Integrator<NumericType>::set_coordinates(std::vector<NumericType> x) {
    for(size_t i=0; i < x.size(); i++) {
        // std::cout << "SC: " << x[i] << std::endl;
    }
    gpuErrchk(hipMemcpy(d_x_t_, &x[0], N_*3*sizeof(NumericType), hipMemcpyHostToDevice));
};

template<typename NumericType> 
void Integrator<NumericType>::set_velocities(std::vector<NumericType> v) {
    for(size_t i=0; i < v.size(); i++) {
        // std::cout << "SV: " << v[i] << std::endl;
    }
    gpuErrchk(hipMemcpy(d_v_t_, &v[0], N_*3*sizeof(NumericType), hipMemcpyHostToDevice));
};

// dangerous! not exception safe.
template<typename NumericType> 
void Integrator<NumericType>::step_cpu(
    const NumericType *h_grads,
    const NumericType *h_hessians,
    const NumericType *h_mixed_partials) {

    gpuErrchk(hipMemcpy(d_grads_, h_grads, N_*3*sizeof(NumericType), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_hessians_, h_hessians, N_*3*N_*3*sizeof(NumericType), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_mixed_partials_, h_mixed_partials, P_*N_*3*sizeof(NumericType), hipMemcpyHostToDevice));

    step_gpu(d_grads_, d_hessians_, d_mixed_partials_);

    hipDeviceSynchronize();
}

template<typename NumericType> 
void Integrator<NumericType>::step_gpu(
    const NumericType *d_grads,
    const NumericType *d_hessians,
    NumericType *d_mixed_partials) {

    size_t tpb = 32;

    if(d_hessians != nullptr && d_mixed_partials != nullptr) {
        hessian_vector_product(d_hessians_, d_dxdp_t_, d_mixed_partials);
        size_t n_blocks = (P_*N_*3 + tpb - 1) / tpb;

        update_derivatives<NumericType><<<n_blocks, tpb>>>(
            coeff_a_,
            d_coeff_bs_,
            d_mixed_partials_,
            d_dxdp_t_,
            d_dvdp_t_,
            dt_,
            P_*N_*3
        );
        gpuErrchk(hipPeekAtLastError());
    }

    size_t n_blocks = (N_*3 + tpb - 1) / tpb;

    // generate new random numbers
    curandErrchk(templateCurandNormal(cr_rng_, d_rng_buffer_, N_*3, 0.0, 1.0));
    reduce_velocities<NumericType><<<n_blocks, tpb>>>(
        d_rng_buffer_,
        coeff_a_,
        d_coeff_bs_,
        d_coeff_cs_,
        d_grads_,
        dt_,
        d_x_t_,
        d_v_t_,
        N_*3);


    gpuErrchk(hipPeekAtLastError());

    step_ += 1;

}

template<typename NumericType> 
void Integrator<NumericType>::hessian_vector_product(
    const NumericType *d_A,
    NumericType *d_B,
    NumericType *d_C) {

    NumericType alpha = 1.0;
    NumericType beta  = 1.0;
 
    const size_t N3 = N_*3;

    // this is set to UPPER because of fortran ordering
    cublasErrchk(templateSymm(cb_handle_,
        HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER,
        N3, P_,
        &alpha,
        d_A, N3,
        d_B, N3,
        &beta,
        d_C, N3));

}

}

template class timemachine::Integrator<double>;
template class timemachine::Integrator<float>;