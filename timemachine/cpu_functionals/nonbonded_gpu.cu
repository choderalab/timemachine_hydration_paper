#include "hip/hip_runtime.h"
#include "nonbonded_gpu.hpp"
#include "gpu_utils.cuh"


#include <ctime>
#include <iostream>

#define ONE_4PI_EPS0 138.935456

namespace timemachine {



inline __device__ float gpuSqrt(float arg) {
  return sqrtf(arg);
}

inline __device__ double gpuSqrt(double arg) {
  return sqrt(arg);
}

#define HESS_3N3N(i,j,N,di,dj) (di*N*3*N + i*3*N + dj*N + j)
#define HESS_N3N3(i,j,N,di,dj) (i*3*N*3 + di*N*3 + j*3 + dj)
#define HESS_IDX HESS_N3N3

#define N_HARDCODE 2489
#define WARP_SIZE 32

// should we change layout to 3N x 3N to improve coalesced reads and writes?
// probably *especially* important for hessians.

template<typename NumericType>
__global__ void electrostatics_total_derivative(
    const NumericType *coords,
    const NumericType *params, // change to int later?
    const int *global_param_idxs, // change to int later?
    const int *param_idxs,
    const NumericType *scale_matrix,
    NumericType *energy_out,
    NumericType *grad_out,
    NumericType *hessian_out,
    NumericType *mp_out,
    int P,
    int N) {



    const int n_atoms = N;
    // const int N3 = n_atoms*3;

    auto i_idx = blockDim.x*blockIdx.x + threadIdx.x;

    NumericType x0, y0, z0, q0;

    if(i_idx >= n_atoms) {
        x0 = 0.0;
        y0 = 0.0;
        z0 = 0.0;
        q0 = 0.0;
    } else {
        x0 = coords[i_idx*3+0];
        y0 = coords[i_idx*3+1];
        z0 = coords[i_idx*3+2];
        q0 = params[param_idxs[i_idx]];
    }

    NumericType grad_dx = 0;
    NumericType grad_dy = 0;
    NumericType grad_dz = 0;

    NumericType hess_xx = 0;
    NumericType hess_yx = 0;
    NumericType hess_yy = 0;
    NumericType hess_zx = 0;
    NumericType hess_zy = 0;
    NumericType hess_zz = 0;

    int num_y_tiles = blockIdx.x + 1;

    for(int tile_y_idx = 0; tile_y_idx < num_y_tiles; tile_y_idx++) {

        NumericType x1, y1, z1, q1;
        NumericType shfl_grad_dx = 0;
        NumericType shfl_grad_dy = 0;
        NumericType shfl_grad_dz = 0;

        NumericType shfl_hess_xx = 0;
        NumericType shfl_hess_yx = 0;
        NumericType shfl_hess_yy = 0;
        NumericType shfl_hess_zx = 0;
        NumericType shfl_hess_zy = 0;
        NumericType shfl_hess_zz = 0;

        // load diagonal elements exactly once, shuffle the rest
        int j_idx = tile_y_idx*WARP_SIZE + threadIdx.x;

        if(j_idx >= n_atoms) {
            x1 = 0.0;
            y1 = 0.0;
            z1 = 0.0;
            q1 = 0.0;
        } else {
            x1 = coords[j_idx*3+0];
            y1 = coords[j_idx*3+1];
            z1 = coords[j_idx*3+2];
            q1 = params[param_idxs[j_idx]];
        }

        // off diagonal
        // iterate over a block of i's
        #pragma unroll 4
        for(int round=0; round < WARP_SIZE; round++) {
            NumericType xi = __shfl_sync(0xffffffff, x0, round);
            NumericType yi = __shfl_sync(0xffffffff, y0, round);
            NumericType zi = __shfl_sync(0xffffffff, z0, round);
            NumericType qi = __shfl_sync(0xffffffff, q0, round);

            int h_i_idx = blockIdx.x*WARP_SIZE + round;
            int h_j_idx = j_idx;

            NumericType dx = xi - x1;
            NumericType dy = yi - y1;
            NumericType dz = zi - z1;
            NumericType d2x = dx*dx;
            NumericType d2y = dy*dy;
            NumericType d2z = dz*dz;

            NumericType d2ij = d2x + d2y + d2z;
            NumericType dij = sqrt(d2ij);
            NumericType d3ij = d2ij*dij;
            NumericType d5ij = d3ij*d2ij;

            NumericType sij = 0;
            if(h_i_idx < n_atoms && h_j_idx < n_atoms) {
                sij = scale_matrix[h_i_idx*n_atoms + h_j_idx];
            } else {
                sij = 0;
            }

            NumericType so4eq01 = sij*ONE_4PI_EPS0*qi*q1;
            NumericType hess_prefactor = so4eq01/d5ij;

            if(h_j_idx < h_i_idx && h_i_idx < n_atoms && h_j_idx < n_atoms) {
                hessian_out[HESS_IDX(h_i_idx, h_j_idx, n_atoms, 0, 0)] += hess_prefactor*(d2ij - 3*d2x);
                hessian_out[HESS_IDX(h_i_idx, h_j_idx, n_atoms, 0, 1)] += -3*hess_prefactor*dx*dy;
                hessian_out[HESS_IDX(h_i_idx, h_j_idx, n_atoms, 0, 2)] += -3*hess_prefactor*dx*dz;
                hessian_out[HESS_IDX(h_i_idx, h_j_idx, n_atoms, 1, 0)] += -3*hess_prefactor*dx*dy;
                hessian_out[HESS_IDX(h_i_idx, h_j_idx, n_atoms, 1, 1)] += hess_prefactor*(d2ij - 3*d2y);
                hessian_out[HESS_IDX(h_i_idx, h_j_idx, n_atoms, 1, 2)] += -3*hess_prefactor*dy*dz;
                hessian_out[HESS_IDX(h_i_idx, h_j_idx, n_atoms, 2, 0)] += -3*hess_prefactor*dx*dz;
                hessian_out[HESS_IDX(h_i_idx, h_j_idx, n_atoms, 2, 1)] += -3*hess_prefactor*dy*dz;
                hessian_out[HESS_IDX(h_i_idx, h_j_idx, n_atoms, 2, 2)] += hess_prefactor*(d2ij - 3*d2z);
            }

        }

        // diagonal elements
        for(int round=0; round < WARP_SIZE; round++) {

            j_idx = tile_y_idx*WARP_SIZE + j_idx % WARP_SIZE;

            NumericType dx = x0 - x1;
            NumericType dy = y0 - y1;
            NumericType dz = z0 - z1;
            NumericType d2x = dx*dx;
            NumericType d2y = dy*dy;
            NumericType d2z = dz*dz;

            NumericType d2ij = d2x + d2y + d2z;
            NumericType dij = sqrt(d2ij);
            NumericType d3ij = d2ij*dij;
            NumericType d5ij = d3ij*d2ij;

            NumericType sij = 0;
            if(i_idx < n_atoms && j_idx < n_atoms) {
                sij = scale_matrix[i_idx*n_atoms + j_idx];
            } else {
                sij = 0;
            }

            NumericType so4eq01 = sij*ONE_4PI_EPS0*q0*q1;
            NumericType grad_prefactor = so4eq01/d3ij;
            NumericType hess_prefactor = so4eq01/d5ij;

            if(j_idx < i_idx && i_idx < n_atoms && j_idx < n_atoms) {

                grad_dx -= grad_prefactor*dx;
                grad_dy -= grad_prefactor*dy;
                grad_dz -= grad_prefactor*dz;

                shfl_grad_dx += grad_prefactor*dx;
                shfl_grad_dy += grad_prefactor*dy;
                shfl_grad_dz += grad_prefactor*dz;

                // compute lower triangular elements
                hess_xx += hess_prefactor*(-d2ij + 3*d2x);
                hess_yx += 3*hess_prefactor*dx*dy;
                hess_yy += hess_prefactor*(-d2ij + 3*d2y);
                hess_zx += 3*hess_prefactor*dx*dz;
                hess_zy += 3*hess_prefactor*dy*dz;
                hess_zz += hess_prefactor*(-d2ij + 3*d2z);

                shfl_hess_xx += hess_prefactor*(-d2ij + 3*d2x);
                shfl_hess_yx += 3*hess_prefactor*dx*dy;
                shfl_hess_yy += hess_prefactor*(-d2ij + 3*d2y);
                shfl_hess_zx += 3*hess_prefactor*dx*dz;
                shfl_hess_zy += 3*hess_prefactor*dy*dz;
                shfl_hess_zz += hess_prefactor*(-d2ij + 3*d2z);

            }

            int srcLane = (threadIdx.x + 1) % WARP_SIZE;

            x1 = __shfl_sync(0xffffffff, x1, srcLane);
            y1 = __shfl_sync(0xffffffff, y1, srcLane);
            z1 = __shfl_sync(0xffffffff, z1, srcLane);
            q1 = __shfl_sync(0xffffffff, q1, srcLane);

            shfl_grad_dx = __shfl_sync(0xffffffff, shfl_grad_dx, srcLane);
            shfl_grad_dy = __shfl_sync(0xffffffff, shfl_grad_dy, srcLane);
            shfl_grad_dz = __shfl_sync(0xffffffff, shfl_grad_dz, srcLane);

            shfl_hess_xx = __shfl_sync(0xffffffff, shfl_hess_xx, srcLane);
            shfl_hess_yx = __shfl_sync(0xffffffff, shfl_hess_yx, srcLane);
            shfl_hess_yy = __shfl_sync(0xffffffff, shfl_hess_yy, srcLane);
            shfl_hess_zx = __shfl_sync(0xffffffff, shfl_hess_zx, srcLane);
            shfl_hess_zy = __shfl_sync(0xffffffff, shfl_hess_zy, srcLane);
            shfl_hess_zz = __shfl_sync(0xffffffff, shfl_hess_zz, srcLane);

            j_idx += 1;

        }

        int target_idx = tile_y_idx*WARP_SIZE + j_idx % WARP_SIZE;

        if(target_idx < n_atoms) {
            atomicAdd(grad_out + target_idx*3 + 0, shfl_grad_dx);
            atomicAdd(grad_out + target_idx*3 + 1, shfl_grad_dy);
            atomicAdd(grad_out + target_idx*3 + 2, shfl_grad_dz);

            atomicAdd(hessian_out + HESS_IDX(target_idx, target_idx, n_atoms, 0, 0), shfl_hess_xx);
            atomicAdd(hessian_out + HESS_IDX(target_idx, target_idx, n_atoms, 1, 0), shfl_hess_yx);
            atomicAdd(hessian_out + HESS_IDX(target_idx, target_idx, n_atoms, 1, 1), shfl_hess_yy);
            atomicAdd(hessian_out + HESS_IDX(target_idx, target_idx, n_atoms, 2, 0), shfl_hess_zx);
            atomicAdd(hessian_out + HESS_IDX(target_idx, target_idx, n_atoms, 2, 1), shfl_hess_zy);
            atomicAdd(hessian_out + HESS_IDX(target_idx, target_idx, n_atoms, 2, 2), shfl_hess_zz);
        }

    }

    if(i_idx < n_atoms) {


        atomicAdd(grad_out + i_idx*3 + 0, grad_dx);
        atomicAdd(grad_out + i_idx*3 + 1, grad_dy);
        atomicAdd(grad_out + i_idx*3 + 2, grad_dz);

        atomicAdd(hessian_out + HESS_IDX(i_idx, i_idx, n_atoms, 0, 0), hess_xx);
        atomicAdd(hessian_out + HESS_IDX(i_idx, i_idx, n_atoms, 1, 0), hess_yx);
        atomicAdd(hessian_out + HESS_IDX(i_idx, i_idx, n_atoms, 1, 1), hess_yy);
        atomicAdd(hessian_out + HESS_IDX(i_idx, i_idx, n_atoms, 2, 0), hess_zx);
        atomicAdd(hessian_out + HESS_IDX(i_idx, i_idx, n_atoms, 2, 1), hess_zy);
        atomicAdd(hessian_out + HESS_IDX(i_idx, i_idx, n_atoms, 2, 2), hess_zz);

    }

};

template <typename NumericType>
ElectrostaticsGPU<NumericType>::ElectrostaticsGPU(
    std::vector<NumericType> params,
    std::vector<size_t> global_param_idxs,
    std::vector<size_t> param_idxs,
    std::vector<NumericType> scale_matrix
) {

    // convert to int version
    std::vector<int> int_global_param_idxs;
    for(auto a : global_param_idxs) {
        int_global_param_idxs.push_back(a);
    }
    std::vector<int> int_param_idxs;
    for(auto a : param_idxs) {
        int_param_idxs.push_back(a);
    }

    gpuErrchk(hipMalloc((void**)&d_params_, params.size()*sizeof(NumericType)));
    gpuErrchk(hipMalloc((void**)&d_global_param_idxs_, int_global_param_idxs.size()*sizeof(*d_global_param_idxs_)));
    gpuErrchk(hipMalloc((void**)&d_param_idxs_, int_param_idxs.size()*sizeof(*d_param_idxs_)));
    gpuErrchk(hipMalloc((void**)&d_scale_matrix_, scale_matrix.size()*sizeof(NumericType)));

    gpuErrchk(hipMemcpy(d_params_, &params[0], params.size()*sizeof(NumericType), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_global_param_idxs_, &int_global_param_idxs[0], int_global_param_idxs.size()*sizeof(*d_global_param_idxs_), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_param_idxs_, &int_param_idxs[0], int_param_idxs.size()*sizeof(*d_param_idxs_), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_scale_matrix_, &scale_matrix[0], scale_matrix.size()*sizeof(NumericType), hipMemcpyHostToDevice));

};


template <typename NumericType>
ElectrostaticsGPU<NumericType>::~ElectrostaticsGPU() {

    gpuErrchk(hipFree(d_params_));
    gpuErrchk(hipFree(d_global_param_idxs_));
    gpuErrchk(hipFree(d_param_idxs_));
    gpuErrchk(hipFree(d_scale_matrix_));

};


template <typename NumericType>
void ElectrostaticsGPU<NumericType>::total_derivative(
    const size_t n_atoms,
    const size_t n_params,
    const NumericType* d_coords, // [N, 3]
    NumericType* d_energy_out, // []
    NumericType* d_grad_out, // [N,3]
    NumericType* d_hessian_out, // [N, 3, N, 3]
    NumericType* d_mp_out // [P, N, 3]
) {

    size_t tpb = 32;
    size_t n_blocks = (n_atoms + tpb - 1) / tpb;

    electrostatics_total_derivative<<<n_blocks, tpb>>>(
        d_coords,
        d_params_, // change to int later?
        d_global_param_idxs_, // change to int later?
        d_param_idxs_,
        d_scale_matrix_,
        d_energy_out,
        d_grad_out,
        d_hessian_out,
        d_mp_out,
        n_params,
        n_atoms);

};


template <typename NumericType>
void ElectrostaticsGPU<NumericType>::total_derivative_cpu(
    const size_t N,
    const size_t P,
    const NumericType* coords, // [N, 3]
    NumericType* energy_out, // []
    NumericType* grad_out, // [N,3]
    NumericType* hessian_out, // [N, 3, N, 3]
    NumericType* mp_out // [P, N, 3]
) {

    NumericType* d_coords; // []
    NumericType* d_energy_out; // []
    NumericType* d_grad_out; // [N,3]
    NumericType* d_hessian_out; // [N, 3, N, 3]
    NumericType* d_mp_out; // [P, N, 3]

    // this is a debugging function.

    gpuErrchk(hipMalloc((void**)&d_coords, N*3*sizeof(NumericType)));
    gpuErrchk(hipMalloc((void**)&d_energy_out, sizeof(NumericType)));
    gpuErrchk(hipMalloc((void**)&d_grad_out, N*3*sizeof(NumericType)));
    gpuErrchk(hipMalloc((void**)&d_hessian_out, N*3*N*3*sizeof(NumericType)));
    gpuErrchk(hipMalloc((void**)&d_mp_out, P*N*3*sizeof(NumericType)));

    gpuErrchk(hipMemcpy(d_coords, coords, N*3*sizeof(NumericType), hipMemcpyHostToDevice));

    std::cout << "CALLING" << std::endl;
    std::clock_t start; double duration; start = std::clock();

    total_derivative(
        N,
        P,
        d_coords,
        d_energy_out,
        d_grad_out,
        d_hessian_out,
        d_mp_out);

    hipDeviceSynchronize();

    duration = ( std::clock() - start ) / (double) CLOCKS_PER_SEC; std::cout<<"ES_DURATION: "<< duration <<'\n';

    gpuErrchk(hipMemcpy(energy_out, d_energy_out, sizeof(NumericType), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(grad_out, d_grad_out, N*3*sizeof(NumericType), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(hessian_out, d_hessian_out, N*3*N*3*sizeof(NumericType), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(mp_out, d_mp_out, P*N*3*sizeof(NumericType), hipMemcpyDeviceToHost));

    gpuErrchk(hipFree(d_coords));
    gpuErrchk(hipFree(d_energy_out));
    gpuErrchk(hipFree(d_grad_out));
    gpuErrchk(hipFree(d_hessian_out));
    gpuErrchk(hipFree(d_mp_out));

};

}

template class timemachine::ElectrostaticsGPU<float>;
template class timemachine::ElectrostaticsGPU<double>;
