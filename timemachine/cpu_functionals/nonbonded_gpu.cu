#include "hip/hip_runtime.h"
#include "nonbonded_gpu.hpp"
#include "gpu_utils.cuh"


#include <ctime>
#include <iostream>

#define ONE_4PI_EPS0 138.935456

namespace timemachine {



inline __device__ float gpuSqrt(float arg) {
  return sqrtf(arg);
}

inline __device__ double gpuSqrt(double arg) {
  return sqrt(arg);
}


#define HESS_3N3N(i,j,N,di,dj) (di*N*3*N + i*3*N + dj*N + j)
#define HESS_N3N3(i,j,N,di,dj) (i*3*N*3 + di*N*3 + j*3 + dj)
// #define HESS_N3N3(i,j,N,di,dj) ((((j)-1)*(ld))+((i)-1))


/*
(ytz): The shuffling nonbonded algorithm

    wi warp index
    ri round i


    w0   |  w1   |  w2    
  0 1 2 3|4 5 6 7|8 9 0 1
  -----------------------
0 0 1 2 3|0 1 2 3|0 1 2 3
1 x 0 1 2|3 0 1 2|3 0 1 2
2 x x 0 1|2 3 0 1|2 3 0 1
3 x x x 0|1 2 3 0|1 2 3 0
- -----------------------
4 x x x x|0 1 2 3|0 1 2 3
5 x x x x|x 0 1 2|3 0 1 2
6 x x x x|x x 0 1|2 3 0 1
7 x x x x|x x x 0|1 2 3 0
- -----------------------
8 x x x x|x x x x|0 1 2 3
9 x x x x|x x x x|x 0 1 2
0 x x x x|x x x x|x x 0 1
1 x x x x|x x x x|x x x 0


t |0 1 2 3|0 1 2 3|0 1 2 3|0 1 2 3
----------------------------------
w0|  r0   |  r1   |  r2   |  r3
--|-------|-------|-------|-------
i |0 1 2 3|0 1 2 3|0 1 2 3|0 1 2 3
j |0 1 2 3|1 2 3 0|2 3 0 1|3 0 1 2


t |0 1 2 3|0 1 2 3|0 1 2 3|0 1 2 3
----------------------------------
w1|  r0   |  r1   |  r2   |  r3
--|-------|-------|-------|-------
i |4 5 6 7|4 5 6 7|4 5 6 7|4 5 6 7
j |0 1 2 3|1 2 3 0|2 3 0 1|3 0 1 2 


warp 1 round 0







*/
#define N_HARDCODE 2489
#define WARP_SIZE 32

// should we change layout to 3N x 3N to improve coalesced reads and writes?
// probably *especially* important for hessians.

template<typename NumericType>
__global__ void electrostatics_total_derivative(
    const NumericType *coords,
    const NumericType *params, // change to int later?
    const int *global_param_idxs, // change to int later?
    const int *param_idxs,
    const NumericType *scale_matrix,
    NumericType *energy_out,
    NumericType *grad_out,
    NumericType *hessian_out,
    NumericType *mp_out,
    int P,
    int N) {

    const int n_atoms = N;
    const int N3 = n_atoms*3;

    auto i_idx = blockDim.x*blockIdx.x + threadIdx.x;

    NumericType x0, y0, z0, q0;

    if(i_idx >= n_atoms) {
        x0 = 0.0;
        y0 = 0.0;
        z0 = 0.0;
        q0 = 0.0;
    } else {
        x0 = coords[i_idx*3+0];
        y0 = coords[i_idx*3+1];
        z0 = coords[i_idx*3+2];
        q0 = params[param_idxs[i_idx]];
    }

    NumericType grad_dx = 0;
    NumericType grad_dy = 0;
    NumericType grad_dz = 0;

    NumericType hess_xx = 0;
    NumericType hess_xy = 0;
    NumericType hess_xz = 0;
    NumericType hess_yx = 0;
    NumericType hess_yy = 0;
    NumericType hess_yz = 0;
    NumericType hess_zx = 0;
    NumericType hess_zy = 0;
    NumericType hess_zz = 0;

    // int tile_x_idx = (n_atoms + WARP_SIZE - 1)/WARP_SIZE;
    // int num_y_tiles = tile_x_idx;

    // int tile_x_idx = (n_atoms + WARP_SIZE - 1)/WARP_SIZE;
    int num_y_tiles = blockIdx.x + 1;

    // printf("block %d NUM_Y_TILES %d\n", blockIdx.x, num_y_tiles);

    for(int tile_y_idx = 0; tile_y_idx < num_y_tiles; tile_y_idx++) {

        NumericType x1, y1, z1, q1;
        NumericType shfl_grad_dx = 0;
        NumericType shfl_grad_dy = 0;
        NumericType shfl_grad_dz = 0;

        NumericType shfl_hess_xx = 0;
        NumericType shfl_hess_xy = 0;
        NumericType shfl_hess_xz = 0;
        NumericType shfl_hess_yx = 0;
        NumericType shfl_hess_yy = 0;
        NumericType shfl_hess_yz = 0;
        NumericType shfl_hess_zx = 0;
        NumericType shfl_hess_zy = 0;
        NumericType shfl_hess_zz = 0;

        // load diagonal elements exactly once, shuffle the rest
        int j_idx = tile_y_idx*WARP_SIZE + threadIdx.x;

        if(j_idx >= n_atoms) {
            x1 = 0.0;
            y1 = 0.0;
            z1 = 0.0;
            q1 = 0.0;
        } else {
            x1 = coords[j_idx*3+0];
            y1 = coords[j_idx*3+1];
            z1 = coords[j_idx*3+2];
            q1 = params[param_idxs[j_idx]];
        }


        // off diagonal

        // iterate over a block of i's
        for(int round=0; round < WARP_SIZE; round++) {
            NumericType xi = __shfl_sync(0xffffffff, x0, round);
            NumericType yi = __shfl_sync(0xffffffff, y0, round);
            NumericType zi = __shfl_sync(0xffffffff, z0, round);
            NumericType qi = __shfl_sync(0xffffffff, q0, round);

            int h_i_idx = blockIdx.x*WARP_SIZE + round;
            int h_j_idx = j_idx;

            NumericType dx = xi - x1;
            NumericType dy = yi - y1;
            NumericType dz = zi - z1;
            NumericType d2x = dx*dx;
            NumericType d2y = dy*dy;
            NumericType d2z = dz*dz;

            NumericType d2ij = d2x + d2y + d2z;
            NumericType dij = sqrt(d2ij);
            NumericType d3ij = d2ij*dij;
            NumericType d5ij = d3ij*d2ij;

            NumericType sij = 0;
            if(h_i_idx < n_atoms && h_j_idx < n_atoms) {
                sij = scale_matrix[h_i_idx*n_atoms + h_j_idx];
            } else {
                sij = 0;
            }

            NumericType so4eq01 = sij*ONE_4PI_EPS0*qi*q1;
            NumericType grad_prefactor = so4eq01/d3ij;
            NumericType hess_prefactor = so4eq01/d5ij;

            if(h_j_idx < h_i_idx && h_i_idx < n_atoms && h_j_idx < n_atoms) {
                const size_t x_dim = 0;
                const size_t y_dim = 1;
                const size_t z_dim = 2;

                NumericType prefactor = hess_prefactor;
                // printf("OFF DIAGONAL HESSIAN %d %d\n:", h_i_idx, h_j_idx);
                // hessian_out[HESS_N3N3(i_idx, j_idx, N, 0, 0)] += prefactor*(d2ij - 3*d2x);
                // hessian_out[HESS_N3N3(i_idx, j_idx, N, 0, 1)] += -3*prefactor*dx*dy;
                // hessian_out[HESS_N3N3(i_idx, j_idx, N, 0, 2)] += -3*prefactor*dx*dz;
                // hessian_out[HESS_N3N3(i_idx, j_idx, N, 1, 0)] += -3*prefactor*dx*dy;
                // hessian_out[HESS_N3N3(i_idx, j_idx, N, 1, 1)] += prefactor*(d2ij - 3*d2y);
                // hessian_out[HESS_N3N3(i_idx, j_idx, N, 1, 2)] += -3*prefactor*dy*dz;
                // hessian_out[HESS_N3N3(i_idx, j_idx, N, 2, 0)] += -3*prefactor*dx*dz;
                // hessian_out[HESS_N3N3(i_idx, j_idx, N, 2, 1)] += -3*prefactor*dy*dz;
                // hessian_out[HESS_N3N3(i_idx, j_idx, N, 2, 2)] += prefactor*(d2ij - 3*d2z);


                hessian_out[h_i_idx * 3 * N * 3 + x_dim * N * 3 + h_j_idx * 3 + x_dim] += prefactor*(d2ij - 3*d2x);
                hessian_out[h_i_idx * 3 * N * 3 + x_dim * N * 3 + h_j_idx * 3 + y_dim] += -3*prefactor*dx*dy;
                hessian_out[h_i_idx * 3 * N * 3 + x_dim * N * 3 + h_j_idx * 3 + z_dim] += -3*prefactor*dx*dz;
                hessian_out[h_i_idx * 3 * N * 3 + y_dim * N * 3 + h_j_idx * 3 + x_dim] += -3*prefactor*dx*dy;
                hessian_out[h_i_idx * 3 * N * 3 + y_dim * N * 3 + h_j_idx * 3 + y_dim] += prefactor*(d2ij - 3*d2y);
                hessian_out[h_i_idx * 3 * N * 3 + y_dim * N * 3 + h_j_idx * 3 + z_dim] += -3*prefactor*dy*dz;
                hessian_out[h_i_idx * 3 * N * 3 + z_dim * N * 3 + h_j_idx * 3 + x_dim] += -3*prefactor*dx*dz;
                hessian_out[h_i_idx * 3 * N * 3 + z_dim * N * 3 + h_j_idx * 3 + y_dim] += -3*prefactor*dy*dz;
                hessian_out[h_i_idx * 3 * N * 3 + z_dim * N * 3 + h_j_idx * 3 + z_dim] += prefactor*(d2ij - 3*d2z);
            }

        }

        j_idx = tile_y_idx*WARP_SIZE + threadIdx.x;

        // on diagonal
        for(int round=0; round < WARP_SIZE; round++) {

            j_idx = tile_y_idx*WARP_SIZE + j_idx % WARP_SIZE;

            NumericType dx = x0 - x1;
            NumericType dy = y0 - y1;
            NumericType dz = z0 - z1;
            NumericType d2x = dx*dx;
            NumericType d2y = dy*dy;
            NumericType d2z = dz*dz;

            NumericType d2ij = d2x + d2y + d2z;
            NumericType dij = sqrt(d2ij);
            NumericType d3ij = d2ij*dij;
            NumericType d5ij = d3ij*d2ij;

            NumericType sij = 0;
            if(i_idx < n_atoms && j_idx < n_atoms) {
                sij = scale_matrix[i_idx*n_atoms + j_idx];
            } else {
                sij = 0;
            }


            NumericType so4eq01 = sij*ONE_4PI_EPS0*q0*q1;
            NumericType grad_prefactor = so4eq01/d3ij;
            NumericType hess_prefactor = so4eq01/d5ij;


            if(j_idx < i_idx && i_idx < n_atoms && j_idx < n_atoms) {

                // printf("computing gradient between i_idx %d j_idx %d\n", i_idx, j_idx);

                grad_dx -= grad_prefactor*dx;
                grad_dy -= grad_prefactor*dy;
                grad_dz -= grad_prefactor*dz;

                shfl_grad_dx += grad_prefactor*dx;
                shfl_grad_dy += grad_prefactor*dy;
                shfl_grad_dz += grad_prefactor*dz;


                NumericType prefactor = hess_prefactor;

                // compute lower triangular
                hess_xx += hess_prefactor*(-d2ij + 3*d2x);
                hess_yx += 3*hess_prefactor*dx*dy;
                hess_yy += hess_prefactor*(-d2ij + 3*d2y);
                hess_zx += 3*hess_prefactor*dx*dz;
                hess_zy += 3*hess_prefactor*dy*dz;
                hess_zz += hess_prefactor*(-d2ij + 3*d2z);

                const size_t x_dim = 0;
                const size_t y_dim = 1;
                const size_t z_dim = 2;

                shfl_hess_xx += hess_prefactor*(-d2ij + 3*d2x);
                shfl_hess_yx += 3*hess_prefactor*dx*dy;
                shfl_hess_yy += hess_prefactor*(-d2ij + 3*d2y);
                shfl_hess_zx += 3*hess_prefactor*dx*dz;
                shfl_hess_zy += 3*hess_prefactor*dy*dz;
                shfl_hess_zz += hess_prefactor*(-d2ij + 3*d2z);

            }

            int srcLane = (threadIdx.x + 1) % WARP_SIZE;

            // printf("tid %d srcLane %d\n", threadIdx.x, srcLane);
            x1 = __shfl_sync(0xffffffff, x1, srcLane);
            y1 = __shfl_sync(0xffffffff, y1, srcLane);
            z1 = __shfl_sync(0xffffffff, z1, srcLane);
            q1 = __shfl_sync(0xffffffff, q1, srcLane);

            shfl_grad_dx = __shfl_sync(0xffffffff, shfl_grad_dx, srcLane);
            shfl_grad_dy = __shfl_sync(0xffffffff, shfl_grad_dy, srcLane);
            shfl_grad_dz = __shfl_sync(0xffffffff, shfl_grad_dz, srcLane);

            shfl_hess_xx = __shfl_sync(0xffffffff, shfl_hess_xx, srcLane);
            shfl_hess_yx = __shfl_sync(0xffffffff, shfl_hess_yx, srcLane);
            shfl_hess_yy = __shfl_sync(0xffffffff, shfl_hess_yy, srcLane);
            shfl_hess_zx = __shfl_sync(0xffffffff, shfl_hess_zx, srcLane);
            shfl_hess_zy = __shfl_sync(0xffffffff, shfl_hess_zy, srcLane);
            shfl_hess_zz = __shfl_sync(0xffffffff, shfl_hess_zz, srcLane);


            j_idx += 1;
            // compute hessians.

        }

        j_idx = tile_y_idx*WARP_SIZE + j_idx % WARP_SIZE;

        // wrap-around pythonic mod %
        // int target_idx = j_idx - 1;
        int target_idx = j_idx;
        // target_idx = (WARP_SIZE + (target_idx % WARP_SIZE)) % WARP_SIZE;

        if(target_idx < n_atoms) {

            // printf("tid %d accumulating to target_idx %d with %f\n", threadIdx.x, target_idx, shfl_grad_dx);

            atomicAdd(grad_out + target_idx*3 + 0, shfl_grad_dx);
            atomicAdd(grad_out + target_idx*3 + 1, shfl_grad_dy);
            atomicAdd(grad_out + target_idx*3 + 2, shfl_grad_dz);

            atomicAdd(hessian_out + target_idx*3*N3 + 0 * N3 + target_idx * 3 + 0, shfl_hess_xx);
            atomicAdd(hessian_out + target_idx*3*N3 + 1 * N3 + target_idx * 3 + 0, shfl_hess_yx);
            atomicAdd(hessian_out + target_idx*3*N3 + 1 * N3 + target_idx * 3 + 1, shfl_hess_yy);
            atomicAdd(hessian_out + target_idx*3*N3 + 2 * N3 + target_idx * 3 + 0, shfl_hess_zx);
            atomicAdd(hessian_out + target_idx*3*N3 + 2 * N3 + target_idx * 3 + 1, shfl_hess_zy);
            atomicAdd(hessian_out + target_idx*3*N3 + 2 * N3 + target_idx * 3 + 2, shfl_hess_zz);
        }

    }

    if(i_idx < n_atoms) {


        atomicAdd(grad_out + i_idx*3 + 0, grad_dx);
        atomicAdd(grad_out + i_idx*3 + 1, grad_dy);
        atomicAdd(grad_out + i_idx*3 + 2, grad_dz);

        atomicAdd(hessian_out + i_idx*3*N3 + 0 * N3 + i_idx * 3 + 0, hess_xx);
        atomicAdd(hessian_out + i_idx*3*N3 + 1 * N3 + i_idx * 3 + 0, hess_yx);
        atomicAdd(hessian_out + i_idx*3*N3 + 1 * N3 + i_idx * 3 + 1, hess_yy);
        atomicAdd(hessian_out + i_idx*3*N3 + 2 * N3 + i_idx * 3 + 0, hess_zx);
        atomicAdd(hessian_out + i_idx*3*N3 + 2 * N3 + i_idx * 3 + 1, hess_zy);
        atomicAdd(hessian_out + i_idx*3*N3 + 2 * N3 + i_idx * 3 + 2, hess_zz);

    }

};

template <typename NumericType>
ElectrostaticsGPU<NumericType>::ElectrostaticsGPU(
    std::vector<NumericType> params,
    std::vector<size_t> global_param_idxs,
    std::vector<size_t> param_idxs,
    std::vector<NumericType> scale_matrix
) {

    // convert to int version
    std::vector<int> int_global_param_idxs;
    for(auto a : global_param_idxs) {
        int_global_param_idxs.push_back(a);
    }
    std::vector<int> int_param_idxs;
    for(auto a : param_idxs) {
        int_param_idxs.push_back(a);
    }

    gpuErrchk(hipMalloc((void**)&d_params_, params.size()*sizeof(NumericType)));
    gpuErrchk(hipMalloc((void**)&d_global_param_idxs_, int_global_param_idxs.size()*sizeof(*d_global_param_idxs_)));
    gpuErrchk(hipMalloc((void**)&d_param_idxs_, int_param_idxs.size()*sizeof(*d_param_idxs_)));
    gpuErrchk(hipMalloc((void**)&d_scale_matrix_, scale_matrix.size()*sizeof(NumericType)));

    gpuErrchk(hipMemcpy(d_params_, &params[0], params.size()*sizeof(NumericType), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_global_param_idxs_, &int_global_param_idxs[0], int_global_param_idxs.size()*sizeof(*d_global_param_idxs_), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_param_idxs_, &int_param_idxs[0], int_param_idxs.size()*sizeof(*d_param_idxs_), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_scale_matrix_, &scale_matrix[0], scale_matrix.size()*sizeof(NumericType), hipMemcpyHostToDevice));

};


template <typename NumericType>
ElectrostaticsGPU<NumericType>::~ElectrostaticsGPU() {

    gpuErrchk(hipFree(d_params_));
    gpuErrchk(hipFree(d_global_param_idxs_));
    gpuErrchk(hipFree(d_param_idxs_));
    gpuErrchk(hipFree(d_scale_matrix_));

};


template <typename NumericType>
void ElectrostaticsGPU<NumericType>::total_derivative(
    const size_t n_atoms,
    const size_t n_params,
    const NumericType* d_coords, // [N, 3]
    NumericType* d_energy_out, // []
    NumericType* d_grad_out, // [N,3]
    NumericType* d_hessian_out, // [N, 3, N, 3]
    NumericType* d_mp_out // [P, N, 3]
) {

    size_t tpb = 32;
    size_t n_blocks = (n_atoms + tpb - 1) / tpb;

    electrostatics_total_derivative<<<n_blocks, tpb>>>(
        d_coords,
        d_params_, // change to int later?
        d_global_param_idxs_, // change to int later?
        d_param_idxs_,
        d_scale_matrix_,
        d_energy_out,
        d_grad_out,
        d_hessian_out,
        d_mp_out,
        n_params,
        n_atoms);

};


template <typename NumericType>
void ElectrostaticsGPU<NumericType>::total_derivative_cpu(
    const size_t N,
    const size_t P,
    const NumericType* coords, // [N, 3]
    NumericType* energy_out, // []
    NumericType* grad_out, // [N,3]
    NumericType* hessian_out, // [N, 3, N, 3]
    NumericType* mp_out // [P, N, 3]
) {

    NumericType* d_coords; // []
    NumericType* d_energy_out; // []
    NumericType* d_grad_out; // [N,3]
    NumericType* d_hessian_out; // [N, 3, N, 3]
    NumericType* d_mp_out; // [P, N, 3]

    // this is a debugging function.

    gpuErrchk(hipMalloc((void**)&d_coords, N*3*sizeof(NumericType)));
    gpuErrchk(hipMalloc((void**)&d_energy_out, sizeof(NumericType)));
    gpuErrchk(hipMalloc((void**)&d_grad_out, N*3*sizeof(NumericType)));
    gpuErrchk(hipMalloc((void**)&d_hessian_out, N*3*N*3*sizeof(NumericType)));
    gpuErrchk(hipMalloc((void**)&d_mp_out, P*N*3*sizeof(NumericType)));

    gpuErrchk(hipMemcpy(d_coords, coords, N*3*sizeof(NumericType), hipMemcpyHostToDevice));

    std::cout << "CALLING" << std::endl;
    std::clock_t start; double duration; start = std::clock();

    total_derivative(
        N,
        P,
        d_coords,
        d_energy_out,
        d_grad_out,
        d_hessian_out,
        d_mp_out);

    hipDeviceSynchronize();

    duration = ( std::clock() - start ) / (double) CLOCKS_PER_SEC; std::cout<<"ES_DURATION: "<< duration <<'\n';

    gpuErrchk(hipMemcpy(energy_out, d_energy_out, sizeof(NumericType), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(grad_out, d_grad_out, N*3*sizeof(NumericType), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(hessian_out, d_hessian_out, N*3*N*3*sizeof(NumericType), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(mp_out, d_mp_out, P*N*3*sizeof(NumericType), hipMemcpyDeviceToHost));

    gpuErrchk(hipFree(d_coords));
    gpuErrchk(hipFree(d_energy_out));
    gpuErrchk(hipFree(d_grad_out));
    gpuErrchk(hipFree(d_hessian_out));
    gpuErrchk(hipFree(d_mp_out));

};

}

template class timemachine::ElectrostaticsGPU<float>;
template class timemachine::ElectrostaticsGPU<double>;
