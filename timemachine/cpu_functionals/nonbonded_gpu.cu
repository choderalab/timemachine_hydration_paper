#include "hip/hip_runtime.h"
#include "nonbonded_gpu.hpp"
#include "electrostatics.cuh"
#include "lennard_jones.cuh"
#include "gpu_utils.cuh"


#include <ctime>
#include <iostream>


namespace timemachine {



template <typename NumericType>
ElectrostaticsGPU<NumericType>::ElectrostaticsGPU(
    std::vector<NumericType> params,
    std::vector<size_t> global_param_idxs,
    std::vector<size_t> param_idxs,
    std::vector<NumericType> scale_matrix
) : P_(params.size()), N_(param_idxs.size()) {

    // convert to int version
    std::vector<int> int_global_param_idxs;
    for(auto a : global_param_idxs) {
        int_global_param_idxs.push_back(a);
    }
    std::vector<int> int_param_idxs;
    for(auto a : param_idxs) {
        int_param_idxs.push_back(a);
    }

    gpuErrchk(hipMalloc((void**)&d_params_, params.size()*sizeof(NumericType)));
    gpuErrchk(hipMalloc((void**)&d_global_param_idxs_, int_global_param_idxs.size()*sizeof(*d_global_param_idxs_)));
    gpuErrchk(hipMalloc((void**)&d_param_idxs_, int_param_idxs.size()*sizeof(*d_param_idxs_)));
    gpuErrchk(hipMalloc((void**)&d_scale_matrix_, scale_matrix.size()*sizeof(NumericType)));

    gpuErrchk(hipMemcpy(d_params_, &params[0], params.size()*sizeof(NumericType), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_global_param_idxs_, &int_global_param_idxs[0], int_global_param_idxs.size()*sizeof(*d_global_param_idxs_), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_param_idxs_, &int_param_idxs[0], int_param_idxs.size()*sizeof(*d_param_idxs_), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_scale_matrix_, &scale_matrix[0], scale_matrix.size()*sizeof(NumericType), hipMemcpyHostToDevice));

};


template <typename NumericType>
ElectrostaticsGPU<NumericType>::~ElectrostaticsGPU() {

    gpuErrchk(hipFree(d_params_));
    gpuErrchk(hipFree(d_global_param_idxs_));
    gpuErrchk(hipFree(d_param_idxs_));
    gpuErrchk(hipFree(d_scale_matrix_));

};


template <typename NumericType>
void ElectrostaticsGPU<NumericType>::total_derivative(
    const size_t n_atoms,
    const size_t n_params,
    const NumericType* d_coords, // [N, 3]
    NumericType* d_energy_out, // []
    NumericType* d_grad_out, // [N,3]
    NumericType* d_hessian_out, // [N, 3, N, 3]
    NumericType* d_mp_out // [P, N, 3]
) {

    size_t tpb = 32;
    size_t n_blocks = (n_atoms + tpb - 1) / tpb;

    electrostatics_total_derivative<<<n_blocks, tpb>>>(
        d_coords,
        d_params_, // change to int later?
        d_global_param_idxs_, // change to int later?
        d_param_idxs_,
        d_scale_matrix_,
        d_energy_out,
        d_grad_out,
        d_hessian_out,
        d_mp_out,
        n_atoms);

};

template <typename NumericType>
void ElectrostaticsGPU<NumericType>::set_params(const std::vector<NumericType> &params) {
    gpuErrchk(hipMemcpy(d_params_, &params[0], params.size()*sizeof(NumericType), hipMemcpyHostToDevice));
}

template <typename NumericType>
std::vector<NumericType> ElectrostaticsGPU<NumericType>::get_params() const {
    std::vector<NumericType> buf(P_);
    gpuErrchk(hipMemcpy(&buf[0], d_params_, P_*sizeof(NumericType), hipMemcpyDeviceToHost));
    return buf;
}

template <typename NumericType>
std::vector<int> ElectrostaticsGPU<NumericType>::get_param_idxs() const {
    std::vector<int> buf(N_);
    gpuErrchk(hipMemcpy(&buf[0], d_param_idxs_, N_*sizeof(int), hipMemcpyDeviceToHost));
    return buf;
}

template <typename NumericType>
void ElectrostaticsGPU<NumericType>::total_derivative_cpu(
    const size_t N,
    const size_t P,
    const NumericType* coords, // [N, 3]
    NumericType* energy_out, // []
    NumericType* grad_out, // [N,3]
    NumericType* hessian_out, // [N, 3, N, 3]
    NumericType* mp_out // [P, N, 3]
) {

    NumericType* d_coords; // []
    NumericType* d_energy_out; // []
    NumericType* d_grad_out; // [N,3]
    NumericType* d_hessian_out; // [N, 3, N, 3]
    NumericType* d_mp_out; // [P, N, 3]

    // this is a debugging function.

    gpuErrchk(hipMalloc((void**)&d_coords, N*3*sizeof(NumericType)));
    gpuErrchk(hipMalloc((void**)&d_energy_out, sizeof(NumericType)));
    gpuErrchk(hipMalloc((void**)&d_grad_out, N*3*sizeof(NumericType)));
    gpuErrchk(hipMalloc((void**)&d_hessian_out, N*3*N*3*sizeof(NumericType)));
    gpuErrchk(hipMalloc((void**)&d_mp_out, P*N*3*sizeof(NumericType)));

    gpuErrchk(hipMemcpy(d_coords, coords, N*3*sizeof(NumericType), hipMemcpyHostToDevice));

    gpuErrchk(hipMemset(d_energy_out, 0, sizeof(NumericType)));
    gpuErrchk(hipMemset(d_grad_out, 0, N*3*sizeof(NumericType)));
    gpuErrchk(hipMemset(d_hessian_out, 0, N*3*N*3*sizeof(NumericType)));
    gpuErrchk(hipMemset(d_mp_out, 0, P*N*3*sizeof(NumericType)));

    hipDeviceSynchronize();
    std::clock_t start; double duration; start = std::clock();

    total_derivative(
        N,
        P,
        d_coords,
        d_energy_out,
        d_grad_out,
        d_hessian_out,
        d_mp_out);

    hipDeviceSynchronize();

    // duration = ( std::clock() - start ) / (double) CLOCKS_PER_SEC; std::cout<<"ES_DURATION: "<< duration <<'\n';

    gpuErrchk(hipMemcpy(energy_out, d_energy_out, sizeof(NumericType), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(grad_out, d_grad_out, N*3*sizeof(NumericType), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(hessian_out, d_hessian_out, N*3*N*3*sizeof(NumericType), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(mp_out, d_mp_out, P*N*3*sizeof(NumericType), hipMemcpyDeviceToHost));

    gpuErrchk(hipFree(d_coords));
    gpuErrchk(hipFree(d_energy_out));
    gpuErrchk(hipFree(d_grad_out));
    gpuErrchk(hipFree(d_hessian_out));
    gpuErrchk(hipFree(d_mp_out));

};

template <typename NumericType>
LennardJonesGPU<NumericType>::LennardJonesGPU(
    std::vector<NumericType> params,
    std::vector<size_t> global_param_idxs,
    std::vector<size_t> param_idxs,
    std::vector<NumericType> scale_matrix
) : P_(params.size()) {

    // convert to int version
    std::vector<int> int_global_param_idxs;
    for(auto a : global_param_idxs) {
        int_global_param_idxs.push_back(a);
    }
    std::vector<int> int_param_idxs;
    for(auto a : param_idxs) {
        int_param_idxs.push_back(a);
    }

    gpuErrchk(hipMalloc((void**)&d_params_, params.size()*sizeof(NumericType)));
    gpuErrchk(hipMalloc((void**)&d_global_param_idxs_, int_global_param_idxs.size()*sizeof(*d_global_param_idxs_)));
    gpuErrchk(hipMalloc((void**)&d_param_idxs_, int_param_idxs.size()*sizeof(*d_param_idxs_)));
    gpuErrchk(hipMalloc((void**)&d_scale_matrix_, scale_matrix.size()*sizeof(NumericType)));

    gpuErrchk(hipMemcpy(d_params_, &params[0], params.size()*sizeof(NumericType), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_global_param_idxs_, &int_global_param_idxs[0], int_global_param_idxs.size()*sizeof(*d_global_param_idxs_), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_param_idxs_, &int_param_idxs[0], int_param_idxs.size()*sizeof(*d_param_idxs_), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_scale_matrix_, &scale_matrix[0], scale_matrix.size()*sizeof(NumericType), hipMemcpyHostToDevice));

};

template <typename NumericType>
LennardJonesGPU<NumericType>::~LennardJonesGPU() {

    gpuErrchk(hipFree(d_params_));
    gpuErrchk(hipFree(d_global_param_idxs_));
    gpuErrchk(hipFree(d_param_idxs_));
    gpuErrchk(hipFree(d_scale_matrix_));

};

template <typename NumericType>
void LennardJonesGPU<NumericType>::set_params(const std::vector<NumericType> &params) {
    gpuErrchk(hipMemcpy(d_params_, &params[0], params.size()*sizeof(NumericType), hipMemcpyHostToDevice));
}


// refactor into interface class
template <typename NumericType>
void LennardJonesGPU<NumericType>::total_derivative(
    const size_t n_atoms,
    const size_t n_params,
    const NumericType* d_coords, // [N, 3]
    NumericType* d_energy_out, // []
    NumericType* d_grad_out, // [N,3]
    NumericType* d_hessian_out, // [N, 3, N, 3]
    NumericType* d_mp_out // [P, N, 3]
) {

    size_t tpb = 32;
    size_t n_blocks = (n_atoms + tpb - 1) / tpb;

    lennard_jones_total_derivative<<<n_blocks, tpb>>>(
        d_coords,
        d_params_, // change to int later?
        d_global_param_idxs_, // change to int later?
        d_param_idxs_,
        d_scale_matrix_,
        d_energy_out,
        d_grad_out,
        d_hessian_out,
        d_mp_out,
        n_atoms);

};

template <typename NumericType>
std::vector<NumericType> LennardJonesGPU<NumericType>::get_params() const {
    std::vector<NumericType> buf(P_);
    gpuErrchk(hipMemcpy(&buf[0], d_params_, P_*sizeof(NumericType), hipMemcpyDeviceToHost));
    return buf;
}

template <typename NumericType>
void LennardJonesGPU<NumericType>::total_derivative_cpu(
    const size_t N,
    const size_t P,
    const NumericType* coords, // [N, 3]
    NumericType* energy_out, // []
    NumericType* grad_out, // [N,3]
    NumericType* hessian_out, // [N, 3, N, 3]
    NumericType* mp_out // [P, N, 3]
) {

    NumericType* d_coords; // []
    NumericType* d_energy_out; // []
    NumericType* d_grad_out; // [N,3]
    NumericType* d_hessian_out; // [N, 3, N, 3]
    NumericType* d_mp_out; // [P, N, 3]

    // this is a debugging function.

    gpuErrchk(hipMalloc((void**)&d_coords, N*3*sizeof(NumericType)));
    gpuErrchk(hipMalloc((void**)&d_energy_out, sizeof(NumericType)));
    gpuErrchk(hipMalloc((void**)&d_grad_out, N*3*sizeof(NumericType)));
    gpuErrchk(hipMalloc((void**)&d_hessian_out, N*3*N*3*sizeof(NumericType)));
    gpuErrchk(hipMalloc((void**)&d_mp_out, P*N*3*sizeof(NumericType)));

    gpuErrchk(hipMemcpy(d_coords, coords, N*3*sizeof(NumericType), hipMemcpyHostToDevice));

    gpuErrchk(hipMemset(d_energy_out, 0, sizeof(NumericType)));
    gpuErrchk(hipMemset(d_grad_out, 0, N*3*sizeof(NumericType)));
    gpuErrchk(hipMemset(d_hessian_out, 0, N*3*N*3*sizeof(NumericType)));
    gpuErrchk(hipMemset(d_mp_out, 0, P*N*3*sizeof(NumericType)));

    hipDeviceSynchronize();
    std::clock_t start; double duration; start = std::clock();


    total_derivative(
        N,
        P,
        d_coords,
        d_energy_out,
        d_grad_out,
        d_hessian_out,
        d_mp_out);

    hipDeviceSynchronize();

    // duration = ( std::clock() - start ) / (double) CLOCKS_PER_SEC; std::cout<<"LJ_DURATION: "<< duration <<'\n';

    gpuErrchk(hipMemcpy(energy_out, d_energy_out, sizeof(NumericType), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(grad_out, d_grad_out, N*3*sizeof(NumericType), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(hessian_out, d_hessian_out, N*3*N*3*sizeof(NumericType), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(mp_out, d_mp_out, P*N*3*sizeof(NumericType), hipMemcpyDeviceToHost));

    gpuErrchk(hipFree(d_coords));
    gpuErrchk(hipFree(d_energy_out));
    gpuErrchk(hipFree(d_grad_out));
    gpuErrchk(hipFree(d_hessian_out));
    gpuErrchk(hipFree(d_mp_out));

};


}

template class timemachine::ElectrostaticsGPU<float>;
template class timemachine::ElectrostaticsGPU<double>;

template class timemachine::LennardJonesGPU<float>;
template class timemachine::LennardJonesGPU<double>;
