#include <cstdio>
#include "bonded_kernels.cuh"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

template<typename RealType>
void harmonic_bond_hmp_gpu(
    const int num_atoms,
    const int num_params,
    const RealType *coords,
    const RealType *params,
    const RealType *dxdps,
    const int num_bonds,
    const int *bond_idxs,
    const int *param_idxs,
    RealType *grads,
    RealType *hmps) {

    RealType* d_coords;
    RealType* d_params;
    RealType* d_dxdps;
    RealType* d_grads;
    RealType* d_hmps;

    int* d_bond_idxs;
    int* d_param_idxs;

    const auto N = num_atoms;
    const auto P = num_params;
    const auto B = num_bonds;
    
    gpuErrchk(hipMalloc((void**)&d_coords, N*3*sizeof(RealType)));
    gpuErrchk(hipMalloc((void**)&d_params, P*sizeof(RealType)));
    gpuErrchk(hipMalloc((void**)&d_dxdps, P*N*3*sizeof(RealType)));
    gpuErrchk(hipMalloc((void**)&d_grads, N*3*sizeof(RealType)));
    gpuErrchk(hipMalloc((void**)&d_hmps, P*N*3*sizeof(RealType)));

    gpuErrchk(hipMemcpy(d_coords, coords, N*3*sizeof(RealType), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_params, params, P*sizeof(RealType), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_dxdps, dxdps, P*N*3*sizeof(RealType), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_grads, grads, N*3*sizeof(RealType), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_hmps, hmps, P*N*3*sizeof(RealType), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc((void**)&d_bond_idxs, B*2*sizeof(*d_bond_idxs)));
    gpuErrchk(hipMalloc((void**)&d_param_idxs, B*2*sizeof(*d_param_idxs)));
    gpuErrchk(hipMemcpy(d_bond_idxs, bond_idxs, B*2*sizeof(*d_bond_idxs), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_param_idxs, param_idxs, B*2*sizeof(*d_param_idxs), hipMemcpyHostToDevice));

    size_t tpb = 32;
    size_t n_blocks = (num_bonds + tpb - 1) / tpb;

    dim3 dimBlock(tpb);
    dim3 dimGrid(n_blocks, num_params); // x, y

    harmonic_bond_hmp<<<dimGrid, dimBlock>>>(
      num_atoms,
      num_params,
      d_coords,
      d_params,
      d_dxdps,
      num_bonds,
      d_bond_idxs,
      d_param_idxs,
      d_grads,
      d_hmps);

    gpuErrchk(hipPeekAtLastError());

    gpuErrchk(hipMemcpy(grads, d_grads, N*3*sizeof(RealType), hipMemcpyDeviceToHost))
    gpuErrchk(hipMemcpy(hmps, d_hmps, P*N*3*sizeof(RealType), hipMemcpyDeviceToHost))

    gpuErrchk(hipFree(d_coords));
    gpuErrchk(hipFree(d_params));
    gpuErrchk(hipFree(d_dxdps));
    gpuErrchk(hipFree(d_grads));
    gpuErrchk(hipFree(d_hmps));

    gpuErrchk(hipFree(d_bond_idxs));
    gpuErrchk(hipFree(d_param_idxs));

};

// instantiate explicitly
template void harmonic_bond_hmp_gpu<float>(
    const int num_atoms,
    const int num_params,
    const float *coords,
    const float *params,
    const float *dxdps,
    const int num_bonds,
    const int *bond_idxs,
    const int *param_idxs,
    float *grads,
    float *hmps);

template void harmonic_bond_hmp_gpu<double>(
    const int num_atoms,
    const int num_params,
    const double *coords,
    const double *params,
    const double *dxdps,
    const int num_bonds,
    const int *bond_idxs,
    const int *param_idxs,
    double *grads,
    double *hmps);