#include "potential.hpp"
#include "kernel_utils.cuh"

namespace timemachine {

template<typename RealType>
void Potential<RealType>::derivatives_host(
    const int num_atoms,
    const int num_params,
    const RealType *h_coords,
    const RealType *h_params,
    const RealType *h_dxdps,
    RealType *h_E,
    RealType *h_dE_dp,
    RealType *h_dE_dx,
    RealType *h_d2E_dxdp) const {

    const auto N = num_atoms;
    const auto P = num_params;

    RealType* d_coords = nullptr;
    RealType* d_params = nullptr;
    RealType* d_dxdps = nullptr;
    RealType* d_E = nullptr;
    RealType* d_dE_dp = nullptr;
    RealType* d_dE_dx = nullptr;
    RealType* d_d2E_dxdp = nullptr;

    gpuErrchk(hipMalloc((void**)&d_coords, N*3*sizeof(RealType)));
    gpuErrchk(hipMalloc((void**)&d_params, P*sizeof(RealType)));

    gpuErrchk(hipMemcpy(d_coords, h_coords, N*3*sizeof(RealType), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_params, h_params, P*sizeof(RealType), hipMemcpyHostToDevice));

    if(h_dxdps != nullptr) {
        gpuErrchk(hipMalloc((void**)&d_dxdps, P*N*3*sizeof(RealType)));
        gpuErrchk(hipMemcpy(d_dxdps, h_dxdps, P*N*3*sizeof(RealType), hipMemcpyHostToDevice));
    }
    if(h_E != nullptr) {
        gpuErrchk(hipMalloc((void**)&d_E, sizeof(RealType)));
    }
    if(h_dE_dp != nullptr) {
        gpuErrchk(hipMalloc((void**)&d_dE_dp, P*sizeof(RealType)));
    }
    if(h_dE_dx != nullptr) {
        gpuErrchk(hipMalloc((void**)&d_dE_dx, N*3*sizeof(RealType)));
    }
    if(h_d2E_dxdp != nullptr) {
        gpuErrchk(hipMalloc((void**)&d_d2E_dxdp, P*N*3*sizeof(RealType)));
    }

    this->derivatives_device(
        N,
        P,
        d_coords,
        d_params,
        d_dxdps,
        d_E,
        d_dE_dp,
        d_dE_dx,
        d_d2E_dxdp
    );

    gpuErrchk(hipPeekAtLastError());

    if(h_E != nullptr) {
        gpuErrchk(hipMemcpy(h_E, d_E, sizeof(RealType), hipMemcpyDeviceToHost));
    }
    if(h_dE_dp != nullptr) {
        gpuErrchk(hipMemcpy(h_dE_dp, d_dE_dp, P*sizeof(RealType), hipMemcpyDeviceToHost));
    }
    if(h_dE_dx != nullptr) {
        gpuErrchk(hipMemcpy(h_dE_dx, d_dE_dx, N*3*sizeof(RealType), hipMemcpyDeviceToHost));
    }
    if(h_d2E_dxdp != nullptr) {
        gpuErrchk(hipMemcpy(h_d2E_dxdp, d_d2E_dxdp, P*N*3*sizeof(RealType), hipMemcpyDeviceToHost));
    }

    hipFree(d_coords);
    hipFree(d_params);
    hipFree(d_dxdps);
    hipFree(d_E);
    hipFree(d_dE_dp);
    hipFree(d_dE_dx);
    hipFree(d_d2E_dxdp);

}

template class Potential<float>;
template class Potential<double>;

}