#include "potential.hpp"
#include "kernel_utils.cuh"

namespace timemachine {

template<typename RealType>
void Potential<RealType>::derivatives_host(
    const int num_atoms,
    const int num_params,
    const RealType *h_coords, // not null
    const RealType *h_params, // not null
    RealType *h_E, // not null
    RealType *h_dE_dx,
    // parameter derivatives
    const RealType *h_dx_dp,
    const int *h_dp_idxs, // not null but can be size zero
    const int num_dp_idxs,
    RealType *h_dE_dp,
    RealType *h_d2E_dxdp) const {

    const auto N = num_atoms;
    const auto P = num_params;
    const auto DP = num_dp_idxs;

    RealType* d_coords = nullptr;
    RealType* d_params = nullptr;
    RealType* d_dx_dp = nullptr;

    int* d_dp_idxs = nullptr;

    RealType* d_E = nullptr;
    RealType* d_dE_dx = nullptr;
    RealType* d_dE_dp = nullptr;
    RealType* d_d2E_dxdp = nullptr;

    gpuErrchk(hipMalloc((void**)&d_coords, N*3*sizeof(RealType)));
    gpuErrchk(hipMalloc((void**)&d_params, P*sizeof(RealType)));

    gpuErrchk(hipMemcpy(d_coords, h_coords, N*3*sizeof(RealType), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_params, h_params, P*sizeof(RealType), hipMemcpyHostToDevice));

    if(num_dp_idxs > 0) {
        // the device function always take in a nullptr if this is of size zero
        gpuErrchk(hipMalloc((void**)&d_dp_idxs, num_dp_idxs*sizeof(int)));
        gpuErrchk(hipMemcpy(d_dp_idxs, h_dp_idxs, num_dp_idxs*sizeof(int), hipMemcpyHostToDevice));
    }

    if(h_dx_dp != nullptr) {
        gpuErrchk(hipMalloc((void**)&d_dx_dp, DP*N*3*sizeof(RealType)));
        gpuErrchk(hipMemcpy(d_dx_dp, h_dx_dp, DP*N*3*sizeof(RealType), hipMemcpyHostToDevice));
    }

    gpuErrchk(hipMalloc((void**)&d_E, sizeof(RealType)));
    gpuErrchk(hipMemset(d_E, 0, sizeof(RealType)));

    if(h_dE_dx != nullptr) {
        gpuErrchk(hipMalloc((void**)&d_dE_dx, N*3*sizeof(RealType)));
        gpuErrchk(hipMemset(d_dE_dx, 0, N*3*sizeof(RealType)));
    }
    if(h_dE_dp != nullptr) {
        gpuErrchk(hipMalloc((void**)&d_dE_dp, DP*sizeof(RealType)));
        gpuErrchk(hipMemset(d_dE_dp, 0, DP*sizeof(RealType)));
    }
    if(h_d2E_dxdp != nullptr) {
        gpuErrchk(hipMalloc((void**)&d_d2E_dxdp, DP*N*3*sizeof(RealType)));
        gpuErrchk(hipMemset(d_d2E_dxdp, 0, DP*N*3*sizeof(RealType)));
    }

    this->derivatives_device(
        N,
        P,
        d_coords,
        d_params,
        d_E, // never null
        d_dE_dx,

        // parameter derivatives
        d_dx_dp,
        d_dp_idxs,
        num_dp_idxs,
        d_dE_dp,
        d_d2E_dxdp
    );

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipMemcpy(h_E, d_E, sizeof(RealType), hipMemcpyDeviceToHost));

    if(h_dE_dx != nullptr) {
        gpuErrchk(hipMemcpy(h_dE_dx, d_dE_dx, N*3*sizeof(RealType), hipMemcpyDeviceToHost));        
    }
    if(h_dE_dp != nullptr) {
        gpuErrchk(hipMemcpy(h_dE_dp, d_dE_dp, DP*sizeof(RealType), hipMemcpyDeviceToHost));
    }
    if(h_d2E_dxdp != nullptr) {
        gpuErrchk(hipMemcpy(h_d2E_dxdp, d_d2E_dxdp, DP*N*3*sizeof(RealType), hipMemcpyDeviceToHost));
    }

    hipFree(d_coords);
    hipFree(d_params);
    hipFree(d_dx_dp);
    hipFree(d_dp_idxs);
    hipFree(d_E);
    hipFree(d_dE_dp);
    hipFree(d_dE_dx);
    hipFree(d_d2E_dxdp);

}

template class Potential<float>;
template class Potential<double>;

}