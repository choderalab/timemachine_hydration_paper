#include "custom_bonded_gpu.hpp"
#include "harmonic_bond_impl.cuh"

#include <ctime>
#include <iostream>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


namespace timemachine {

template <typename RealType>
HarmonicBond<RealType>::HarmonicBond(
    std::vector<int> bond_idxs,
    std::vector<int> param_idxs
) : n_bonds_(bond_idxs.size()/2) {

    gpuErrchk(hipMalloc((void**)&d_param_idxs_, param_idxs.size()*sizeof(*d_param_idxs_)));
    gpuErrchk(hipMalloc((void**)&d_bond_idxs_, bond_idxs.size()*sizeof(*d_bond_idxs_)));
    gpuErrchk(hipMemcpy(d_param_idxs_, &param_idxs[0], param_idxs.size()*sizeof(*d_param_idxs_), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_bond_idxs_, &bond_idxs[0], bond_idxs.size()*sizeof(*d_bond_idxs_), hipMemcpyHostToDevice));

};

template <typename RealType>
HarmonicBond<RealType>::~HarmonicBond() {
    gpuErrchk(hipFree(d_bond_idxs_));
    gpuErrchk(hipFree(d_param_idxs_));
};

template <typename RealType>
void HarmonicBond<RealType>::derivatives_host(
        const int num_atoms,
        const int num_params,
        const RealType *h_coords,
        const RealType *h_params,
        const RealType *h_dxdps,
        RealType *h_E,
        RealType *h_dE_dp,
        RealType *h_dE_dx,
        RealType *h_d2E_dxdp) const {

    const auto N = num_atoms;
    const auto P = num_params;
    const auto B = n_bonds_;

    RealType* d_coords = nullptr;
    RealType* d_params = nullptr;
    RealType* d_dxdps = nullptr;
    RealType* d_E = nullptr;
    RealType* d_dE_dp = nullptr;
    RealType* d_dE_dx = nullptr;
    RealType* d_d2E_dxdp = nullptr;

    gpuErrchk(hipMalloc((void**)&d_coords, N*3*sizeof(RealType)));
    gpuErrchk(hipMalloc((void**)&d_params, P*sizeof(RealType)));

    gpuErrchk(hipMemcpy(d_coords, h_coords, N*3*sizeof(RealType), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_params, h_params, P*sizeof(RealType), hipMemcpyHostToDevice));

    if(h_dxdps != nullptr) {
        gpuErrchk(hipMalloc((void**)&d_dxdps, P*N*3*sizeof(RealType)));
        gpuErrchk(hipMemcpy(d_dxdps, h_dxdps, P*N*3*sizeof(RealType), hipMemcpyHostToDevice));
    }
    if(h_E != nullptr) {
        gpuErrchk(hipMalloc((void**)&d_E, sizeof(RealType)));
    }
    if(h_dE_dp != nullptr) {
        gpuErrchk(hipMalloc((void**)&d_dE_dp, P*sizeof(RealType)));
    }
    if(h_dE_dx != nullptr) {
        gpuErrchk(hipMalloc((void**)&d_dE_dx, N*3*sizeof(RealType)));
    }
    if(h_d2E_dxdp != nullptr) {
        gpuErrchk(hipMalloc((void**)&d_d2E_dxdp, P*N*3*sizeof(RealType)));
    }

    int tpb = 32;
    int n_blocks = (B + tpb - 1) / tpb;
    int dim_y = P;

    // we don't need the other derivatives if we don't need
    // parameter derivatives
    if(d_dE_dp == nullptr && d_d2E_dxdp == nullptr) {
        dim_y = 1;
    }

    dim3 dimBlock(tpb);
    dim3 dimGrid(n_blocks, dim_y); // x, y

    harmonic_bond_derivatives<<<dimGrid, dimBlock>>>(
        N,
        P,
        d_coords,
        d_params,
        d_dxdps,
        B,
        d_bond_idxs_,
        d_param_idxs_,
        d_E,
        d_dE_dp,
        d_dE_dx,
        d_d2E_dxdp
    );

    gpuErrchk(hipPeekAtLastError());

    if(h_E != nullptr) {
        gpuErrchk(hipMemcpy(h_E, d_E, sizeof(RealType), hipMemcpyDeviceToHost));
    }
    if(h_dE_dp != nullptr) {
        gpuErrchk(hipMemcpy(h_dE_dp, d_dE_dp, P*sizeof(RealType), hipMemcpyDeviceToHost));
    }
    if(h_dE_dx != nullptr) {
        gpuErrchk(hipMemcpy(h_dE_dx, d_dE_dx, N*3*sizeof(RealType), hipMemcpyDeviceToHost));
    }
    if(h_d2E_dxdp != nullptr) {
        gpuErrchk(hipMemcpy(h_d2E_dxdp, d_d2E_dxdp, P*N*3*sizeof(RealType), hipMemcpyDeviceToHost));
    }

    hipFree(d_coords);
    hipFree(d_params);
    hipFree(d_dxdps);
    hipFree(d_E);
    hipFree(d_dE_dp);
    hipFree(d_dE_dx);
    hipFree(d_d2E_dxdp);

};

template class HarmonicBond<float>;
template class HarmonicBond<double>;

} // namespace timemachine