#include <stdexcept>

#include "custom_bonded_gpu.hpp"
#include "k_harmonic_bond.cuh"
#include "kernel_utils.cuh"

namespace timemachine {

template <typename RealType>
HarmonicBond<RealType>::HarmonicBond(
    std::vector<int> bond_idxs,
    std::vector<int> param_idxs
) : n_bonds_(bond_idxs.size()/2) {

    gpuErrchk(hipMalloc((void**)&d_param_idxs_, param_idxs.size()*sizeof(*d_param_idxs_)));
    gpuErrchk(hipMalloc((void**)&d_bond_idxs_, bond_idxs.size()*sizeof(*d_bond_idxs_)));
    gpuErrchk(hipMemcpy(d_param_idxs_, &param_idxs[0], param_idxs.size()*sizeof(*d_param_idxs_), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_bond_idxs_, &bond_idxs[0], bond_idxs.size()*sizeof(*d_bond_idxs_), hipMemcpyHostToDevice));

};

template <typename RealType>
HarmonicBond<RealType>::~HarmonicBond() {
    gpuErrchk(hipFree(d_bond_idxs_));
    gpuErrchk(hipFree(d_param_idxs_));
};

template <typename RealType>
void HarmonicBond<RealType>::derivatives_device(
        const int num_atoms,
        const int num_params,
        const RealType *d_coords,
        const RealType *d_params,
        RealType *d_E,
        RealType *d_dE_dx,

        const RealType *d_dx_dp,
        const int *d_dp_idxs,
        const int num_dp_idxs,
        RealType *d_dE_dp,
        RealType *d_d2E_dxdp) const {

    const auto N = num_atoms;
    const auto P = num_params;
    const auto B = n_bonds_;

    int tpb = 32;
    int n_blocks = (B + tpb - 1) / tpb;
    int dim_y = 1;

    // zero dimension dim_ys are *not* allowed.
    if(num_dp_idxs == 0) {
        // inference mode
        dim_y = 1;
        if(d_dp_idxs != nullptr) {
            throw std::runtime_error("d_dp_idxs is not null but num_dp_idxs == 0!");
        }
    } else {
        dim_y = num_dp_idxs;
    }

    dim3 dimBlock(tpb);
    dim3 dimGrid(n_blocks, dim_y); // x, y

    // if timing, be sure check only this and call hipDeviceSynchronize() after
    // also: ensure compiling with -O2 to get standard optimizations
    harmonic_bond_derivatives<<<dimGrid, dimBlock>>>(
        N,
        P,
        d_coords,
        d_params,
        B,
        d_bond_idxs_,
        d_param_idxs_,
        d_E,
        d_dE_dx,
        // parameter derivatives
        d_dx_dp,
        d_dp_idxs,
        d_dE_dp,
        d_d2E_dxdp
    );

    gpuErrchk(hipPeekAtLastError());

};

template class HarmonicBond<float>;
template class HarmonicBond<double>;

} // namespace timemachine