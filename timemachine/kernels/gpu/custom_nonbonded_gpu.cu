#include <stdexcept>

#include "potential.hpp"
#include "custom_nonbonded_gpu.hpp"
#include "k_lennard_jones.cuh"
#include "kernel_utils.cuh"

#include <chrono>  // for high_resolution_clock
#include <iostream>
namespace timemachine {

template <typename RealType>
LennardJones<RealType>::LennardJones(
    std::vector<RealType> scale_matrix,
    std::vector<int> param_idxs
) {

    gpuErrchk(hipMalloc((void**)&d_param_idxs_, param_idxs.size()*sizeof(*d_param_idxs_)));
    gpuErrchk(hipMalloc((void**)&d_scale_matrix_, scale_matrix.size()*sizeof(*d_scale_matrix_)));
    gpuErrchk(hipMemcpy(d_param_idxs_, &param_idxs[0], param_idxs.size()*sizeof(*d_param_idxs_), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_scale_matrix_, &scale_matrix[0], scale_matrix.size()*sizeof(*d_scale_matrix_), hipMemcpyHostToDevice));

};

template <typename RealType>
LennardJones<RealType>::~LennardJones() {
    gpuErrchk(hipFree(d_param_idxs_));
    gpuErrchk(hipFree(d_scale_matrix_));
};


template <typename RealType>
void LennardJones<RealType>::derivatives_device(
    const int num_confs,
    const int num_atoms,
    const int num_params,
    const RealType *d_coords,
    const RealType *d_params,
    RealType *d_E,
    RealType *d_dE_dx,
    // parameter derivatives
    const RealType *d_dx_dp,
    const int *d_dp_idxs,
    const int num_dp_idxs,
    RealType *d_dE_dp,
    RealType *d_d2E_dxdp) const {


    const auto C = num_confs;
    const auto N = num_atoms;
    const auto P = num_params;

    int tpb = 32;
    int n_blocks = (num_atoms + tpb - 1) / tpb;
    int dim_y = 1;

    // zero dimension dim_ys are *not* allowed.
    if(num_dp_idxs == 0) {
        // inference mode
        dim_y = 1;
        if(d_dp_idxs != nullptr) {
            throw std::runtime_error("d_dp_idxs is not null but num_dp_idxs == 0");
        }
    } else {
        dim_y = num_dp_idxs;
    }

    dim3 dimBlock(tpb);
    dim3 dimGrid(n_blocks, dim_y, C); // x, y, z dims

    auto start = std::chrono::high_resolution_clock::now();
    k_lennard_jones<<<dimGrid, dimBlock>>>(
        N,
        P,
        d_coords,
        d_params,
        d_scale_matrix_,
        d_param_idxs_,
        d_E,
        d_dE_dx,
        // parameter derivatives
        d_dx_dp,
        d_dp_idxs,
        d_dE_dp,
        d_d2E_dxdp
    );
    hipDeviceSynchronize();
    auto finish = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = finish - start;
    std::cout << "LJ Elapsed time: " << elapsed.count() << " s\n";

    gpuErrchk(hipPeekAtLastError());

};

template class LennardJones<float>;
template class LennardJones<double>;

}